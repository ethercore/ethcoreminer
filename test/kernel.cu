#include "hip/hip_runtime.h"
// Inner loop for prog_seed 600
__device__ __forceinline__ void progPowLoop(const uint32_t loop,
    uint32_t mix[PROGPOW_REGS],
    const dag_t *g_dag,
    const uint32_t c_dag[PROGPOW_CACHE_WORDS],
    const bool hack_false)
{
    dag_t data_dag;
    uint32_t offset, data;
    const uint32_t lane_id = threadIdx.x & (PROGPOW_LANES - 1);
    // global load
    offset = __shfl_sync(0xFFFFFFFF, mix[0], loop%PROGPOW_LANES, PROGPOW_LANES);
    offset %= PROGPOW_DAG_ELEMENTS;
    offset = offset * PROGPOW_LANES + (lane_id ^ loop) % PROGPOW_LANES;
    data_dag = g_dag[offset];
    // hack to prevent compiler from reordering LD and usage
    if (hack_false) __threadfence_block();
    // cache load 0
    offset = mix[26] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[0] = (mix[0] ^ data) * 33;
    // random math 0
    data = mix[10] ^ mix[16];
    mix[4] = ROTL32(mix[4], 27) ^ data;
    // cache load 1
    offset = mix[30] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[27] = ROTR32(mix[27], 7) ^ data;
    // random math 1
    data = mix[24] & mix[14];
    mix[26] = (mix[26] * 33) + data;
    // cache load 2
    offset = mix[1] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[13] = (mix[13] * 33) + data;
    // random math 2
    data = mix[17] & mix[16];
    mix[15] = ROTR32(mix[15], 12) ^ data;
    // cache load 3
    offset = mix[19] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[17] = (mix[17] ^ data) * 33;
    // random math 3
    data = mul_hi(mix[31], mix[5]);
    mix[7] = (mix[7] ^ data) * 33;
    // cache load 4
    offset = mix[11] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[14] = (mix[14] ^ data) * 33;
    // random math 4
    data = mix[23] * mix[19];
    mix[8] = (mix[8] * 33) + data;
    // cache load 5
    offset = mix[21] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[9] = (mix[9] ^ data) * 33;
    // random math 5
    data = clz(mix[30]) + clz(mix[15]);
    mix[12] = ROTR32(mix[12], 16) ^ data;
    // cache load 6
    offset = mix[15] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[3] = ROTR32(mix[3], 27) ^ data;
    // random math 6
    data = clz(mix[12]) + clz(mix[5]);
    mix[10] = (mix[10] * 33) + data;
    // cache load 7
    offset = mix[18] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[1] = ROTR32(mix[1], 6) ^ data;
    // random math 7
    data = min(mix[4], mix[25]);
    mix[11] = ROTR32(mix[11], 27) ^ data;
    // cache load 8
    offset = mix[3] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[6] = (mix[6] ^ data) * 33;
    // random math 8
    data = mul_hi(mix[18], mix[16]);
    mix[16] = (mix[16] ^ data) * 33;
    // cache load 9
    offset = mix[17] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[28] = ROTL32(mix[28], 17) ^ data;
    // random math 9
    data = ROTL32(mix[15], mix[23]);
    mix[31] = (mix[31] * 33) + data;
    // cache load 10
    offset = mix[31] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[2] = (mix[2] * 33) + data;
    // random math 10
    data = mix[11] | mix[17];
    mix[19] = ROTL32(mix[19], 28) ^ data;
    // cache load 11
    offset = mix[16] % PROGPOW_CACHE_WORDS;
    data = c_dag[offset];
    mix[30] = ROTR32(mix[30], 18) ^ data;
    // random math 11
    data = mix[22] * mix[7];
    mix[22] = ROTR32(mix[22], 30) ^ data;
    // random math 12
    data = mix[27] & mix[16];
    mix[29] = ROTR32(mix[29], 25) ^ data;
    // random math 13
    data = ROTL32(mix[11], mix[0]);
    mix[5] = (mix[5] ^ data) * 33;
    // random math 14
    data = ROTR32(mix[15], mix[25]);
    mix[24] = ROTL32(mix[24], 13) ^ data;
    // random math 15
    data = mix[14] & mix[26];
    mix[18] = (mix[18] * 33) + data;
    // random math 16
    data = mix[28] * mix[16];
    mix[25] = (mix[25] ^ data) * 33;
    // random math 17
    data = mix[11] * mix[0];
    mix[23] = (mix[23] ^ data) * 33;
    // random math 18
    data = mix[2] + mix[24];
    mix[21] = ROTR32(mix[21], 20) ^ data;
    // random math 19
    data = mix[25] + mix[4];
    mix[20] = ROTL32(mix[20], 22) ^ data;
    // consume global load data
    // hack to prevent compiler from reordering LD and usage
    if (hack_false) __threadfence_block();
    mix[0] = (mix[0] ^ data_dag.s[0]) * 33;
    mix[0] = ROTR32(mix[0], 21) ^ data_dag.s[1];
    mix[4] = (mix[4] * 33) + data_dag.s[2];
    mix[27] = (mix[27] ^ data_dag.s[3]) * 33;
}